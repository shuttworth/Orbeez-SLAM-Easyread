/**
* This file is part of ORB-SLAM2.
*
* Copyright (C) 2014-2016 Raúl Mur-Artal <raulmur at unizar dot es> (University of Zaragoza)
* For more information see <https://github.com/raulmur/ORB_SLAM2>
*
* ORB-SLAM2 is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* ORB-SLAM2 is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with ORB-SLAM2. If not, see <http://www.gnu.org/licenses/>.
*/


#include<iostream>
#include<algorithm>
#include<fstream>
#include<chrono>
#include <sys/wait.h>

#include<opencv2/core/core.hpp>
#include<opencv2/imgcodecs/legacy/constants_c.h>

#include<System.h>

using namespace std;

void LoadImages(const string &strAssociationFilename, vector<string> &vstrImageFilenamesRGB,
                vector<string> &vstrImageFilenamesD, vector<double> &vTimestamps);
string GetDatasetName(const string &strSequencePath);

int main(int argc, char **argv)
{
    if(argc != 5)
    {
        cerr << endl << "Usage: ./rgbd_tum path_to_vocabulary path_to_settings path_to_sequence path_to_association" << endl;
        return 1;
    }

    // Retrieve paths to images
    vector<string> vstrImageFilenamesRGB;
    vector<string> vstrImageFilenamesD;
    vector<double> vTimestamps;
    string strAssociationFilename = string(argv[4]);
    LoadImages(strAssociationFilename, vstrImageFilenamesRGB, vstrImageFilenamesD, vTimestamps);

    // Check consistency in the number of images and depthmaps
    int nImages = vstrImageFilenamesRGB.size();
    if(vstrImageFilenamesRGB.empty())
    {
        cerr << endl << "No images found in provided path." << endl;
        return 1;
    }
    else if(vstrImageFilenamesD.size()!=vstrImageFilenamesRGB.size())
    {
        cerr << endl << "Different number of images for rgb and depth." << endl;
        return 1;
    }

    // Create SLAM system. It initializes all system threads and gets ready to process frames.
    ORBEEZ::System SLAM(argv[1],argv[2],ORBEEZ::System::RGBD, true, false);

    // Vector for tracking time statistics
    vector<float> vTimesTrack;
    vTimesTrack.resize(nImages);

    cout << endl << "-------" << endl;
    cout << "Start processing sequence ..." << endl;
    cout << "Images in the sequence: " << nImages << endl << endl;

    // Main loop
    cv::Mat imRGB, imD;
    for(int ni=0; ni<nImages; ni++)
    {
        // Read image and depthmap from file
        imRGB = cv::imread(string(argv[3])+"/"+vstrImageFilenamesRGB[ni],CV_LOAD_IMAGE_UNCHANGED);
        imD = cv::imread(string(argv[3])+"/"+vstrImageFilenamesD[ni],CV_LOAD_IMAGE_UNCHANGED);
        double tframe = vTimestamps[ni];

        if(imRGB.empty())
        {
            cerr << endl << "Failed to load image at: "
                 << string(argv[3]) << "/" << vstrImageFilenamesRGB[ni] << endl;
            return 1;
        }


        std::chrono::steady_clock::time_point t1 = std::chrono::steady_clock::now();

        // Pass the image to the SLAM system
        SLAM.TrackRGBD(imRGB,imD,tframe);

        std::chrono::steady_clock::time_point t2 = std::chrono::steady_clock::now();

        double ttrack= std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count();

        vTimesTrack[ni]=ttrack;

        // Wait to load the next frame
        double T=0;
        if(ni<nImages-1)
            T = vTimestamps[ni+1]-tframe;
        else if(ni>0)
            T = tframe-vTimestamps[ni-1];

        if(ttrack<T)
            usleep((T-ttrack)*1e6);
    }

    // Stop orb-viewer and tracking. 
    // The user can watch the Nerf screen
    SLAM.Spin();

    // Tracking time statistics
    sort(vTimesTrack.begin(),vTimesTrack.end());
    float totaltime = 0;
    for(int ni=0; ni<nImages; ni++)
    {
        totaltime+=vTimesTrack[ni];
    }
    cout << "-------" << endl << endl;
    cout << "median tracking time: " << vTimesTrack[nImages/2] << endl;
    cout << "mean tracking time: " << totaltime/nImages << endl;

    string dataset_name = GetDatasetName(string(argv[3])); 
    auto trajString = "evaluation/RGBD_TUM_"+dataset_name+"_KeyFrameTrajectory";
    auto snapString = "evaluation/RGBD_TUM_"+dataset_name+".msgpack";
    auto gtJsonTrajString = "evaluation/RGBD_TUM_"+dataset_name+"_gtTraj.json";

    // Save camera trajectory
    SLAM.SaveTrajectoryTUM("evaluation/RGBD_TUM_"+dataset_name+"_CameraTrajectory.txt");
    SLAM.SaveKeyFrameTrajectoryTUM(trajString+".txt");  // rpj only
    SLAM.SaveKeyFrameTrajectoryNGP(trajString+".json"); // rpj (+ pht if train extrinsics) 
    SLAM.SaveSnapShot(snapString);

    int pid = fork();
    if (pid < 0)
    {
        cout << "fork failed" << endl;
    }
    else if( pid == 0 )
    {
        // For headless version, we do not need to spin the program.
        // But instead, terminate training process and execute evaluation script.
        auto gtString = string(argv[3]) + "/groundtruth.txt";
        auto trajPathString = trajString + ".txt";
        auto plotString = trajString + "_rpj.png";
        char *gtPath = (char *)(gtString.c_str());
        char *trajPath = (char *)(trajPathString.c_str());
        char *plotPath = (char *)(plotString.c_str());
        char *gtJsonTrajPath = (char *)(gtJsonTrajString.c_str());

        std::cout << "ATE w/ reprojection error:" << std::endl;
        char *execArgs[] = {"python3", "scripts/evaluate_ate.py", gtPath, trajPath, "--verbose", "--plot", plotPath, "--save_gt_json", gtJsonTrajPath, NULL};
        execvp("python3", execArgs);
    }
    wait(NULL);

    std::cout << std::endl;

    pid = fork();
    if (pid < 0)
    {
        cout << "fork failed" << endl;
    }
    else if( pid == 0 )
    {
        // For headless version, we do not need to spin the program.
        // But instead, terminate training process and execute evaluation script.
        auto gtString = string(argv[3]) + "/groundtruth.txt";
        auto trajPathString = trajString + ".json";
        auto plotString = trajString + "_rpj+pht.png";
        char *gtPath = (char *)(gtString.c_str());
        char *trajPath = (char *)(trajPathString.c_str());
        char *plotPath = (char *)(plotString.c_str());

        std::cout << "ATE w/ reprojection error (+ photometric error if optimize extrinsic == true):" << std::endl;
        char *execArgs[] = {"python3", "scripts/evaluate_ate.py", gtPath, trajPath, "--verbose", "--plot", plotPath, NULL};
        execvp("python3", execArgs);
    }
    wait(NULL);

#ifdef ORBEEZ_GUI
    SLAM.AddGroundTruthTraj(gtJsonTrajString);

    cout << "Press ctrl + c to exit the program " << endl;

    // Don't stop program, to see the Nerf training result
    volatile int keep_spinning = 1;
    while (keep_spinning) ; // spin
#endif        

    return 0;
}

void LoadImages(const string &strAssociationFilename, vector<string> &vstrImageFilenamesRGB,
                vector<string> &vstrImageFilenamesD, vector<double> &vTimestamps)
{
    ifstream fAssociation;
    fAssociation.open(strAssociationFilename.c_str());
    while(!fAssociation.eof())
    {
        string s;
        getline(fAssociation,s);
        if(!s.empty())
        {
            stringstream ss;
            ss << s;
            double t;
            string sRGB, sD;
            ss >> t;
            vTimestamps.push_back(t);
            ss >> sRGB;
            vstrImageFilenamesRGB.push_back(sRGB);
            ss >> t;
            ss >> sD;
            vstrImageFilenamesD.push_back(sD);

        }
    }

}

string GetDatasetName(const string &strSequencePath) 
{
    string s(strSequencePath);
    std::string delimiter = "/";

    size_t pos = 0;
    std::string token;
    while ((pos = s.find(delimiter)) != std::string::npos) {
        token = s.substr(0, pos);
        s.erase(0, pos + delimiter.length());
    }

    if (s.length() == 0)
        return token;
    else
        return s;
}