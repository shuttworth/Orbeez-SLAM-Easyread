#include "hip/hip_runtime.h"
/**
* This file is part of ORB-SLAM2.
*
* Copyright (C) 2014-2016 Raúl Mur-Artal <raulmur at unizar dot es> (University of Zaragoza)
* For more information see <https://github.com/raulmur/ORB_SLAM2>
*
* ORB-SLAM2 is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* ORB-SLAM2 is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with ORB-SLAM2. If not, see <http://www.gnu.org/licenses/>.
*/

#include "Map.h"   // IWYU pragma: associated

#include "KeyFrame.h"
#include "MapPoint.h"

#include "opencv2/core/eigen.hpp"
#include <unistd.h>

#include <neural-graphics-primitives/common.h> 

#include<mutex>

namespace Eigen
{
    // reference from https://github.com/nlohmann/json/issues/3267
    template<typename Scalar, int Rows, int Cols>
    void to_json(nlohmann::json& j, const Matrix<Scalar, Rows, Cols>& matrix) {
        for (int row = 0; row < matrix.rows(); ++row) {
            nlohmann::json column = nlohmann::json::array();
            for (int col = 0; col < matrix.cols(); ++col) {
                column.push_back(matrix(row, col));
            }
            j.push_back(column);
        }
    }

    template<typename Scalar, int Rows, int Cols>
    void from_json(const nlohmann::json& j, Matrix<Scalar, Rows, Cols>& matrix) {        
        for (std::size_t row = 0; row < j.size(); ++row) {
            const auto& jrow = j.at(row);
            for (std::size_t col = 0; col < jrow.size(); ++col) {
                const auto& value = jrow.at(col);
                value.get_to(matrix(row, col));
            }
        }
    }
}

namespace ORBEEZ
{

// Map类的构造函数，主要用于初始化地图相关的参数和设置
Map::Map(const string &strSettingPath, const string &strSlamTransform, const bool bTrainCameraWithPhotometric):mnMaxKFid(0), mnBigChangeIdx(0), mbDataIsReady(false)
{
    mpTestbed = std::make_shared<ngp::Testbed>(ngp::ETestbedMode::NerfSlam);
    cv::FileStorage fSettings(strSettingPath, cv::FileStorage::READ);

    // Must exist
    // 解析配置文件中的相机参数 (焦距、畸变系数、图像尺寸)
    cv::FileNode camera_node = fSettings["Camera"];
    float fl_x                        = KeyCheck(camera_node, "fx");
    float fl_y                        = KeyCheck(camera_node, "fy");
    float k1                          = KeyCheck(camera_node, "k1");
    float k2                          = KeyCheck(camera_node, "k2");
    float p1                          = KeyCheck(camera_node, "p1");
    float p2                          = KeyCheck(camera_node, "p2");
    float cx                          = KeyCheck(camera_node, "cx");
    float cy                          = KeyCheck(camera_node, "cy");
    int width                         = KeyCheck(camera_node, "width");
    int height                        = KeyCheck(camera_node, "height");
    cv::FileNode NeRF_node   = fSettings["NeRF"];
    // 解析配置文件中的 NeRF 网络参数 (AABB 尺度、缩放、偏移)
    int aabb_scale                    = KeyCheck(NeRF_node, "aabb_scale");
    float scale                       = KeyCheck(NeRF_node, "scale");
    cv::FileNode offset               = KeyCheck(NeRF_node, "offset");
    std::string network_config_path   = KeyCheck(NeRF_node, "network_config_path");

    std::vector<float> offset_vector;
    for(cv::FileNodeIterator it = offset.begin(); it != offset.end(); it++)
    {
        offset_vector.push_back((float)*it);
    }
    // nlohmann::json offset_array = nlohmann::json::parse(offset_vector.begin(), offset_vector.end());
    
    // Only if RGB-D
    cv::FileNode depth_node           = fSettings["DepthMapFactor"];

    if (!depth_node.empty())
    {
        float DepthMapFactor = fSettings["DepthMapFactor"];
        // Although is name as integer, instant-ngp requires float.
        m_scene_config["integer_depth_scale"] = 1.0f/DepthMapFactor;
    }

    m_scene_config["fl_x"]                  = fl_x;
    m_scene_config["fl_y"]                  = fl_y;
    m_scene_config["k1"]                    = k1;
    m_scene_config["k2"]                    = k2;
    m_scene_config["p1"]                    = p1;
    m_scene_config["p2"]                    = p2;
    m_scene_config["cx"]                    = cx;
    m_scene_config["cy"]                    = cy;
    m_scene_config["w"]                     = width;
    m_scene_config["h"]                     = height;
    m_scene_config["aabb_scale"]            = aabb_scale;
    m_scene_config["scale"]                 = scale;
    m_scene_config["offset"]                = offset_vector;

    std::string json_string =  m_scene_config.dump(4);
    std::cout << json_string << std::endl;

    std::ofstream ofs(strSlamTransform, std::ofstream::trunc);
    ofs << json_string << std::endl;
    ofs.close();

    mpTestbed->load_training_data(strSlamTransform);
    mpTestbed->reload_network_from_file(network_config_path);
    mpTestbed->m_train = true;
    mpTestbed->m_nerf.training.optimize_extrinsics = bTrainCameraWithPhotometric;
}

cv::FileNode Map::KeyCheck(cv::FileNode& parent_node, const string& key) 
{
    cv::FileNode node = parent_node[key];
    if (node.empty()) 
    {
        throw runtime_error{string{"The key: "} + key + string{" is not in the yaml. "}};
    }
    else{
        return node;
    }
}

void Map::init_window()
{
    unique_lock<mutex> lock(mMutexMap);
#ifdef ORBEEZ_GUI
    mpTestbed->init_window(1920, 1080);
#endif
}

void Map::StopTraining()
{
    mpTestbed->m_train = false;
}

// 函数作用：主要用于处理单帧地图数据并进行训练和渲染(在NeRF数据准备好的情况下)
bool Map::frame()
{
    // The lock ensure modifying dataset (add keyframe) and training won't do simultaneously
    unique_lock<mutex> lock(mMutexMap);

    if (mbDataIsReady){

        // Draw sparse point cloud
        // 生成稀疏点云
        const vector<MapPoint*> &vpMPs    = vector<MapPoint*>(mspMapPoints.begin(),mspMapPoints.end());
        const vector<MapPoint*> &vpRefMPs = mvpReferenceMapPoints;
        set<MapPoint*> spRefMPs(vpRefMPs.begin(), vpRefMPs.end());

        std::vector<Eigen::Vector3f> map_points;
        for(size_t i=0, iend=vpMPs.size(); i<iend;i++)
        {
            if(vpMPs[i]->isBad() || spRefMPs.count(vpMPs[i]))
                continue;
            cv::Mat pos = vpMPs[i]->GetWorldPos();
            Eigen::Vector3f map_point;
            cv::cv2eigen(pos, map_point);
            map_points.push_back(map_point);
        }

        std::vector<Eigen::Vector3f> ref_map_points;
        for(set<MapPoint*>::iterator sit=spRefMPs.begin(), send=spRefMPs.end(); sit!=send; sit++)
        {
            if((*sit)->isBad())
                continue;
            cv::Mat pos = (*sit)->GetWorldPos();
            Eigen::Vector3f ref_map_point;
            cv::cv2eigen(pos, ref_map_point);
            ref_map_points.push_back(ref_map_point);
        }

        // 添加稀疏点云到测试环境
        mpTestbed->add_sparse_point_cloud(map_points, ref_map_points);

        // train and render  训练和渲染
        bool value = mpTestbed->frame();

        if(mpTestbed->m_train)
            tlog::info() << "iteration=" << mpTestbed->m_training_step << " loss=" << mpTestbed->m_loss_scalar.val();

        return value;
    }
    else
        return false;
}

Eigen::Matrix<float, 3, 4> Map::KeyFrameWorldPoseToNGPFormat(const Eigen::Matrix<float, 3, 4>& slam_matrix) const
{
    return mpTestbed->m_nerf.training.dataset.slam_matrix_to_ngp(slam_matrix);
}

Eigen::Matrix<float, 3, 4> Map::KeyFrameNGPFormatToWorldPose(const Eigen::Matrix<float, 3, 4>& ngp_matrix) const
{
    return mpTestbed->m_nerf.training.dataset.ngp_matrix_to_slam(ngp_matrix);
}

Eigen::Matrix<float, 3, 4> Map::PoseWithPhotometric(int index) const
{
    return KeyFrameNGPFormatToWorldPose(mpTestbed->m_nerf.training.transforms[index].start);
}

void Map::AddKeyFrame(KeyFrame *pKF)
{
    unique_lock<mutex> lock(mMutexMap);
    mspKeyFrames.insert(pKF);
    mId2KeyFrame[pKF->mnId] = pKF;

    if(pKF->mnId>mnMaxKFid)
        mnMaxKFid=pKF->mnId;

    // if (mspKeyFrames.size() < 5)
    // if (pKF->mnId % 5 == 0) {
    Eigen::MatrixXf transform_matrix;
    mbDataIsReady = true;
    cv::cv2eigen(pKF->GetPoseInverse(), transform_matrix);

    nlohmann::json frame_config;
    frame_config["Id"]                        = pKF->mnId;  
    frame_config["fl_x"]                      = m_scene_config["fl_x"];
    frame_config["fl_y"]                      = m_scene_config["fl_y"];
    frame_config["transform_matrix"]          = transform_matrix;
    frame_config["w"]                         = m_scene_config["w"];
    frame_config["h"]                         = m_scene_config["h"];

    cv::Mat color_image                       = pKF->GetColorImage();
    cv::Mat depth_image                       = pKF->GetDepthImage();

    // std::cout << "[Map] frame_config:" << frame_config.dump(4) << std::endl;

    if (frame_config["w"] != color_image.cols || frame_config["h"] != color_image.rows)
    {
        throw runtime_error("Image size does not match the yaml. Maybe use the wrong yaml");
    }

    // instant ngp requires image to have 4 channels
    cv::Mat rgba;
    if (color_image.channels() == 3) {
        // https://stackoverflow.com/questions/32290096/python-opencv-add-alpha-channel-to-rgb-image
        // First create the image with alpha channel
        cv::cvtColor(color_image, rgba, cv::COLOR_BGR2RGBA);

        // Split the image for access to alpha channel
        cv::Mat channels[4];
        cv::split(rgba, channels);

        // Assign the mask to the last channel of the image
        channels[3] = 255 * cv::Mat::ones(channels[3].rows, channels[3].cols, CV_8UC1);

        // Finally concat channels for rgba image
        cv::merge(channels, 4, rgba);
    }
    else if (color_image.channels() == 4){
        // instant-ngp may release the image memory. Therefore, clone the image.
        rgba = color_image.clone();
    }
    else{
        throw std::runtime_error("incorrect image format");
    }

    assert(rgba.isContinuous());

    // prepare an additional memory for instant-ngp. It will be free in instant-ngp
    uint8_t *img = (uint8_t*)malloc(sizeof(uint8_t) * rgba.rows * rgba.cols * rgba.channels());
    memcpy(img, rgba.data, sizeof(uint8_t) * rgba.rows * rgba.cols * rgba.channels());

    uint16_t *depth = nullptr;
    
    // if(!depth_image.empty()){

    //     if (frame_config["w"] != depth_image.cols || frame_config["h"] != depth_image.rows)
    //     {
    //         throw runtime_error("depth image must be same as color_image. Please resize them in the main program");
    //     }

    //     depth = (uint16_t*)malloc(sizeof(uint16_t) * depth_image.rows * depth_image.cols * depth_image.channels());
    //     memcpy(depth, depth_image.data, sizeof(uint8_t) * depth_image.rows * depth_image.cols * depth_image.channels());
    // }

    std::tuple<ngp::TrainingXForm*,int> t = mpTestbed->add_training_image(frame_config, img, depth);
    ngp::TrainingXForm *pXform = std::get<0>(t);
    int index = std::get<1>(t);

    pKF->SetNerfXformPointer(pXform, index);
}

void Map::update_transformsGPU()
{
    unique_lock<mutex> lock(mMutexMap);
    mpTestbed->update_camera(mpTestbed->m_training_stream);
}

bool Map::NerfCameraIsUpdated()
{
    unique_lock<mutex> lock(mMutexMap);
    return (mpTestbed->m_nerf.training.optimize_extrinsics) && (mpTestbed->m_nerf.training.n_steps_since_cam_update == 0);
}

void Map::GetCameraInfo(float *slice_plane_z, float *scale, float *fov, float *dof)
{
    *slice_plane_z = mpTestbed->m_slice_plane_z;
    *scale = mpTestbed->scale();
    *fov = mpTestbed->fov();
    *dof = mpTestbed->m_dof;
}

nlohmann::json Map::GetSceneConfig()
{
    return m_scene_config;
}

void Map::CullEmptyRegion()
{
    unique_lock<mutex> lock(mMutexMap);
    std::cout << "Cull empty region (camera can't see) in density grid" << std::endl;
    // Mesh use inference stream
    mpTestbed->cull_empty_region(false, mpTestbed->m_inference_stream);
}

void Map::SaveSnapShot(const string &filename)
{
    unique_lock<mutex> lock(mMutexMap);
    mpTestbed->save_snapshot(filename, false);
}

void Map::SaveMesh(const string &filename, uint32_t marching_cubes_res)
{
    unique_lock<mutex> lock(mMutexMap);
    // std::cout << "SaveMesh" << std::endl;
    Eigen::Vector3i res3d(3);
    res3d << marching_cubes_res, marching_cubes_res, marching_cubes_res;
    mpTestbed->compute_and_save_marching_cubes_mesh(filename.c_str(), res3d);
}

void Map::AddGroundTruthTraj(const std::string& gtTrajPath)
{
    unique_lock<mutex> lock(mMutexMap);
    mpTestbed->AddGroundTruthTraj(gtTrajPath);
}

void Map::AddMapPoint(MapPoint *pMP)
{
    unique_lock<mutex> lock(mMutexMap);
    mspMapPoints.insert(pMP);
}

void Map::EraseMapPoint(MapPoint *pMP)
{
    unique_lock<mutex> lock(mMutexMap);
    mspMapPoints.erase(pMP);

    // TODO: This only erase the pointer.
    // Delete the MapPoint
}

void Map::EraseKeyFrame(KeyFrame *pKF)
{
    unique_lock<mutex> lock(mMutexMap);
    mId2KeyFrame.erase(pKF->mnId);
    mspKeyFrames.erase(pKF);

    // TODO: This only erase the pointer.
    // Delete the MapPoint
}

void Map::SetReferenceMapPoints(const vector<MapPoint *> &vpMPs)
{
    unique_lock<mutex> lock(mMutexMap);
    mvpReferenceMapPoints = vpMPs;
}

void Map::InformNewBigChange()
{
    unique_lock<mutex> lock(mMutexMap);
    mnBigChangeIdx++;
}

int Map::GetLastBigChangeIdx()
{
    unique_lock<mutex> lock(mMutexMap);
    return mnBigChangeIdx;
}

vector<KeyFrame*> Map::GetAllKeyFrames()
{
    unique_lock<mutex> lock(mMutexMap);
    return vector<KeyFrame*>(mspKeyFrames.begin(),mspKeyFrames.end());
}

vector<MapPoint*> Map::GetAllMapPoints()
{
    unique_lock<mutex> lock(mMutexMap);
    return vector<MapPoint*>(mspMapPoints.begin(),mspMapPoints.end());
}

long unsigned int Map::MapPointsInMap()
{
    unique_lock<mutex> lock(mMutexMap);
    return mspMapPoints.size();
}

long unsigned int Map::KeyFramesInMap()
{
    unique_lock<mutex> lock(mMutexMap);
    return mspKeyFrames.size();
}

vector<MapPoint*> Map::GetReferenceMapPoints()
{
    unique_lock<mutex> lock(mMutexMap);
    return mvpReferenceMapPoints;
}

long unsigned int Map::GetMaxKFid()
{
    unique_lock<mutex> lock(mMutexMap);
    return mnMaxKFid;
}

void Map::clear()
{
    mpTestbed->clear_training_data();
    mpTestbed->load_nerfslam();

    for(set<MapPoint*>::iterator sit=mspMapPoints.begin(), send=mspMapPoints.end(); sit!=send; sit++)
        delete *sit;

    for(set<KeyFrame*>::iterator sit=mspKeyFrames.begin(), send=mspKeyFrames.end(); sit!=send; sit++)
        delete *sit;

    mspMapPoints.clear();
    mspKeyFrames.clear();
    mId2KeyFrame.clear();
    mnMaxKFid = 0;
    mvpReferenceMapPoints.clear();
    mvpKeyFrameOrigins.clear();
}

} //namespace ORBEEZ
